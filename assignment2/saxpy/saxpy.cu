#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int n = sizeof(float);
    int totalBytes = n * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc((void **)&device_x, n * N);
    hipMalloc((void **)&device_y, n * N);
    hipMalloc((void **)&device_result, n * N);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, n * N, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, n * N, hipMemcpyHostToDevice);

    // run kernel
    double startComputationTime = CycleTimer::currentSeconds();

    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();

    double endComputationTime = CycleTimer::currentSeconds();
    printf("Computation time: %.3lfms\n", (endComputationTime - startComputationTime) * 1000.);
    //
    // TODO copy result from GPU using hipMemcpy
    //
    hipMemcpy(resultarray, device_result, n * N, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // TODO free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
